/* 
 * File:        Parameters.cu
 * Author:      Jiri Jaros
 * Affiliation: Brno University of Technology
 *              Faculty of Information Technology
 *              
 *              and
 * 
 *              The Australian National University
 *              ANU College of Engineering & Computer Science
 *
 * Email:       jarosjir@fit.vutbr.cz
 * Web:         www.fit.vutbr.cz/~jarosjir
 * 
 * Comments:    Implementation file of the parameter class. 
 *              This class maintains all the parameters of evolution.
 *
 * 
 * License:     This source code is distribute under OpenSouce GNU GPL license
 *                
 *              If using this code, please consider citation of related papers
 *              at http://www.fit.vutbr.cz/~jarosjir/pubs.php        
 *      
 *
 * 
 * Created on 30 March 2012, 00:00 PM
 * Modified on 17 February 2017, 15:59
 */


#include <iostream>
#include <getopt.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "Parameters.h"

//----------------------------------------------------------------------------//
//                              Definitions                                   //
//----------------------------------------------------------------------------//

__constant__  TEvolutionParameters GPU_EvolutionParameters;


// Singleton initialization 
bool TParameters::pTParametersInstanceFlag = false;
TParameters* TParameters::pTParametersSingle = NULL;


//----------------------------------------------------------------------------//
//                              Implementation                                //
//                              public methods                                //
//----------------------------------------------------------------------------//

/*
 * Get instance of TPrarams
 */
TParameters* TParameters::GetInstance(){
    if(! pTParametersInstanceFlag)
    {        
        pTParametersSingle = new TParameters();
        pTParametersInstanceFlag = true;
        return pTParametersSingle;
    }
    else
    {
        return pTParametersSingle;
    }
}// end of TParameters::GetInstance
//-----------------------------------------------------------------------------

void TParameters::LoadParameters(const int& popsize, const int& maxEvals,
    const float& mutationRate, const float& crossRate,
    const int& statsInterval, const string& filename, const string& instance) {
    float OffspringPercentage = 0.5f;
    EvolutionParameters.PopulationSize = popsize;
    EvolutionParameters.MaxEvaluations = maxEvals;
    EvolutionParameters.MutationPst = mutationRate;
    EvolutionParameters.CrossoverPst = crossRate;
    EvolutionParameters.StatisticsInterval = statsInterval;
    outputFilename = filename;
    GlobalDataFileName = instance;
    // Set population size to be even.
   EvolutionParameters.OffspringPopulationSize = (int) (OffspringPercentage * EvolutionParameters.PopulationSize);
   if (EvolutionParameters.OffspringPopulationSize == 0) EvolutionParameters.OffspringPopulationSize = 2;
   if (EvolutionParameters.OffspringPopulationSize % 2 == 1) EvolutionParameters.OffspringPopulationSize++;
   
      
   // Set UINT mutation threshold to faster comparison
   EvolutionParameters.MutationUINTBoundary  = (unsigned int) ((float) UINT_MAX * EvolutionParameters.MutationPst);
   EvolutionParameters.CrossoverUINTBoundary = (unsigned int) ((float) UINT_MAX * EvolutionParameters.CrossoverPst);
   
}
void TParameters::LoadParameters(const int& popsize, const int& maxEvals,
    const float& mutationRate, const float& crossRate,
    const int& statsInterval, const string& filename) {
    float OffspringPercentage = 0.5f;
    EvolutionParameters.PopulationSize = popsize;
    EvolutionParameters.MaxEvaluations = maxEvals;
    EvolutionParameters.MutationPst = mutationRate;
    EvolutionParameters.CrossoverPst = crossRate;
    EvolutionParameters.StatisticsInterval = statsInterval;
    outputFilename = filename;
    // Set population size to be even.
   EvolutionParameters.OffspringPopulationSize = (int) (OffspringPercentage * EvolutionParameters.PopulationSize);
   if (EvolutionParameters.OffspringPopulationSize == 0) EvolutionParameters.OffspringPopulationSize = 2;
   if (EvolutionParameters.OffspringPopulationSize % 2 == 1) EvolutionParameters.OffspringPopulationSize++;
   
      
   // Set UINT mutation threshold to faster comparison
   EvolutionParameters.MutationUINTBoundary  = (unsigned int) ((float) UINT_MAX * EvolutionParameters.MutationPst);
   EvolutionParameters.CrossoverUINTBoundary = (unsigned int) ((float) UINT_MAX * EvolutionParameters.CrossoverPst);
   
}


/*
 * Load parameters from command line
 * 
 * @param argc
 * @param argv
 * 
 */
void TParameters::LoadParametersFromCommandLine(int argc, char **argv){
    
   // default values
   float OffspringPercentage = 0.5f;
   const int num_args = 8;
   if (argc != num_args) {
       std::cerr << "Error." << std::endl;
       PrintUsageAndExit();
   } else {
       EvolutionParameters.PopulationSize = atoi(argv[1]);
       EvolutionParameters.MaxEvaluations = atoi(argv[2]);
       EvolutionParameters.MutationPst = atof(argv[3]);
       EvolutionParameters.CrossoverPst = atof(argv[4]);
       EvolutionParameters.StatisticsInterval = atoi(argv[5]);
       outputFilename = argv[6];
       GlobalDataFileName = argv[7];
   }

   // Set population size to be even.
   EvolutionParameters.OffspringPopulationSize = (int) (OffspringPercentage * EvolutionParameters.PopulationSize);
   if (EvolutionParameters.OffspringPopulationSize == 0) EvolutionParameters.OffspringPopulationSize = 2;
   if (EvolutionParameters.OffspringPopulationSize % 2 == 1) EvolutionParameters.OffspringPopulationSize++;
   
      
   // Set UINT mutation threshold to faster comparison
   EvolutionParameters.MutationUINTBoundary  = (unsigned int) ((float) UINT_MAX * EvolutionParameters.MutationPst);
   EvolutionParameters.CrossoverUINTBoundary = (unsigned int) ((float) UINT_MAX * EvolutionParameters.CrossoverPst);
   
} // end of LoadParametersFromCommandLine
//------------------------------------------------------------------------------


/*
 * Copy parameters to the GPU constant memory
 */
void TParameters::StoreParamsOnGPU(){
            
    checkCudaErrors(
        hipMemcpyToSymbol(HIP_SYMBOL(GPU_EvolutionParameters), &EvolutionParameters, sizeof(TEvolutionParameters) )
    );
    
   
}// end of StoreParamsOnGPU
//------------------------------------------------------------------------------


//----------------------------------------------------------------------------//
//                              Implementation                                //
//                              private methods                               //
//----------------------------------------------------------------------------//

/*
 * Constructor
 */
TParameters::TParameters(){
    
    EvolutionParameters.PopulationSize      = 128;
    EvolutionParameters.ChromosomeSize      = 32;
    // Definido a 400K tras un experimento previo
    EvolutionParameters.MaxEvaluations      = 500000;
        
    EvolutionParameters.MutationPst         = 0.01f;
    EvolutionParameters.CrossoverPst        = 0.7f;    
    EvolutionParameters.OffspringPopulationSize = (int) (0.5f * EvolutionParameters.PopulationSize);
    
    EvolutionParameters.StatisticsInterval  = 1;
    
    EvolutionParameters.IntBlockSize        = sizeof(int)*8;  
    GlobalDataFileName                      = "";
    
    FPrintBest                              = false;
    
    // New parameters
}// end of TParameters
//------------------------------------------------------------------------------

/*
 * print usage of the algorithm
 */
void TParameters::PrintUsageAndExit(){
    
  cerr << "Usage: " << endl;
  cerr << "./gpu_knapsack popsize max_evals mutation_rate cross_rate";
  cerr << " stats_internal output_filename instance_filename" << endl;  

  cerr << endl;
  cerr << "Default Population_size       = 128"  << endl;
  cerr << "Default Number_of_generations = 100" << endl;
  cerr << endl;
  
  cerr << "Default mutation_rate  = 0.01" << endl;
  cerr << "Default crossover_rate = 0.7" << endl;
  cerr << "Default offspring_rate = 0.5" << endl;
  cerr << endl;
  

  cerr << "Default statistics_interval = 1"   << endl;
  
  cerr << "Default benchmark_file_name = knapsack_data.txt\n";
  
  exit(EXIT_FAILURE);
    
}// end of PrintUsage
//------------------------------------------------------------------------------





/*
 * Print all parameters
 * 
 */
void TParameters::PrintAllParameters(){
    printf("-----------------------------------------\n");
    printf("--- Evolution parameters --- \n");
    printf("Population size:     %d\n", EvolutionParameters.PopulationSize);
    printf("Offspring size:      %d\n", EvolutionParameters.OffspringPopulationSize);
    printf("Chromosome int size: %d\n", EvolutionParameters.ChromosomeSize);
    printf("Chromosome size:     %d\n", EvolutionParameters.ChromosomeSize * EvolutionParameters.IntBlockSize);
    
    printf("Max Evaluations:  %d\n", EvolutionParameters.MaxEvaluations);
    printf("\n");
        
    
    printf("Crossover pst:       %f\n", EvolutionParameters.CrossoverPst);
    printf("Mutation  pst:       %f\n", EvolutionParameters.MutationPst);
    printf("Crossover  int:      %u\n",EvolutionParameters.CrossoverUINTBoundary);    
    printf("Mutation  int:       %u\n", EvolutionParameters.MutationUINTBoundary);    
    printf("\n");
    
    printf("Statistics interval: %d\n", EvolutionParameters.StatisticsInterval);
    
    printf("\n");
    printf("Data File: %s\n",GlobalDataFileName.c_str());
    printf("-----------------------------------------\n");
    
    
}// end of PrintAllParameters
//------------------------------------------------------------------------------