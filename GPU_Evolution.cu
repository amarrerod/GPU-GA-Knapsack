/* 
 * File:        GPU_Evolution.cu
 * Author:      Jiri Jaros
 * Affiliation: Brno University of Technology
 *              Faculty of Information Technology
 *              
 *              and
 * 
 *              The Australian National University
 *              ANU College of Engineering & Computer Science
 *
 * Email:       jarosjir@fit.vutbr.cz
 * Web:         www.fit.vutbr.cz/~jarosjir
 * 
 * Comments:    Implementation file of the GA evolution
 *              This class controls the evolution process on multicore CPU
 *
 * 
 * License:     This source code is distribute under OpenSouce GNU GPL license
 *                
 *              If using this code, please consider citation of related papers
 *              at http://www.fit.vutbr.cz/~jarosjir/pubs.php        
 *      
 *
 * 
 * Created on 30 March 2012, 00:00 PM
 */

#include <iostream>
#include <stdio.h>
#include <sys/time.h>

#include "GPU_Evolution.h"
#include "GPU_Statistics.h"
#include "CUDA_Kernels.h"
#include "Parameters.h"

using namespace std;


//----------------------------------------------------------------------------//
//                              Definitions                                   //
//----------------------------------------------------------------------------//


//----------------------------------------------------------------------------//
//                              Implementation                                //
//                              public methods                                //
//----------------------------------------------------------------------------//

/*
 * Constructor of the class
 */
TGPU_Evolution::TGPU_Evolution(){
    
    // Create parammeter class
    Params    = TParameters::GetInstance();
    pDeviceIdx = 0;
    
    // Select device 
    hipSetDevice(pDeviceIdx);
    CheckAndReportCudaError(__FILE__,__LINE__);    

    // Get parameters of the device
    hipDeviceProp_t 	prop;	
    hipGetDeviceProperties (&prop, pDeviceIdx);
    CheckAndReportCudaError(__FILE__,__LINE__);
   
    pMultiprocessorCount = prop.multiProcessorCount;
    Params->SetGPU_SM_Count(prop.multiProcessorCount);
    
    
    // Load data from file
    GlobalData.LoadFromFile();
    
    // Create populations on GPU
    MasterPopulation    = new TGPU_Population(Params->PopulationSize(), Params->ChromosomeSize());    
    OffspringPopulation = new TGPU_Population(Params->OffspringPopulationSize(), Params->ChromosomeSize());
            
    // Create statistics
    GPUStatistics = new TGPU_Statistics();
    pActGeneration = 0;    
    performedEvaluations = 0;
    // Initialize Random seed
    InitSeed();
    
}// end of TGPU_Evolution
//------------------------------------------------------------------------------
  

/*
 * Destructor of the class
 */
TGPU_Evolution::~TGPU_Evolution(){
    delete MasterPopulation;    
    delete OffspringPopulation;
    delete GPUStatistics;
}// end of Destructor
//------------------------------------------------------------------------------

/*
 * Run Evolution
 */
void TGPU_Evolution::Run(){
    Initialize();
    RunEvolutionCycle();        
}// end of Run
//------------------------------------------------------------------------------

/*
 * Initialize seed
 */
void TGPU_Evolution::InitSeed() {
  struct timeval tp1;  
  gettimeofday(&tp1, NULL);  
  pSeed = (tp1.tv_sec / (pDeviceIdx + 1)) * tp1.tv_usec;
};// end of InitSeed
//------------------------------------------------------------------------------

/*
 * Initialization of the GA
 */
void TGPU_Evolution::Initialize(){
           
    pActGeneration = 0;
        
    // Store parameters on GPU and print them out
    Params->StoreParamsOnGPU();
    Params->PrintAllParameters();
    
    // Set elements count
       
    int Elements = Params->ChromosomeSize() * Params->PopulationSize();
                            
    //-- Initialize population --//
    FirstPopulationGenerationKernel
            <<<pMultiprocessorCount * 2, BLOCK_SIZE>>>
            (MasterPopulation->DeviceData, GetSeed());
    CheckAndReportCudaError(__FILE__,__LINE__);

    dim3 Blocks; 
    dim3 Threads;
            
    Blocks.x = 1;    
    Blocks.y = (Params->PopulationSize() / (CHR_PER_BLOCK) +1);
    Blocks.z = 1;
    
    
    Threads.x = WARP_SIZE;
    Threads.y = CHR_PER_BLOCK;
    Threads.z = 1;   
    
    // Calculate Knapsack fintess
    CalculateKnapsackFintess
            <<<Blocks, Threads>>>
                (MasterPopulation->DeviceData, GlobalData.DeviceData);
    CheckAndReportCudaError(__FILE__,__LINE__);
     
}// end of TGPU_Evolution
//------------------------------------------------------------------------------




/*
 * Run evolutionary cycle for defined number of generations
 *
 * CHANGES: Run evolutionary cycle until reaching Max evaluations
 */
void TGPU_Evolution::RunEvolutionCycle(){
    outputFile.open(Params->OutputFilename());
    if(!outputFile.is_open()){
        cerr << "Error while trying to open: " << Params->OutputFilename() << endl;
        exit(-1);
    } 
    dim3 Blocks;
    dim3 Threads;
    
    Threads.x = WARP_SIZE;
    Threads.y = CHR_PER_BLOCK;
    Threads.z = 1;

    // Evaluate generations
    //for (pActGeneration = 1; pActGeneration < Params->NumOfGenerations(); pActGeneration++) {
    for(performedEvaluations = 0; performedEvaluations < Params->MaxEvaluations(); 
        performedEvaluations += Params->PopulationSize()) {
          //-------------Selection -----------//
          Blocks.x = 1;    
          Blocks.y = (Params->OffspringPopulationSize() % (CHR_PER_BLOCK << 1)  == 0) ?
                            Params->OffspringPopulationSize() / (CHR_PER_BLOCK << 1)  :
                            Params->OffspringPopulationSize() / (CHR_PER_BLOCK << 1) + 1;

          Blocks.z = 1;

          GeneticManipulationKernel
                  <<<Blocks, Threads>>>
                  (MasterPopulation->DeviceData, OffspringPopulation->DeviceData, GetSeed());
          CheckAndReportCudaError(__FILE__,__LINE__);

          //----------- Evaluation ---------//

          Blocks.x = 1;    
          Blocks.y = (Params->OffspringPopulationSize() % (CHR_PER_BLOCK)  == 0) ?
                            Params->OffspringPopulationSize() / (CHR_PER_BLOCK)  :
                            Params->OffspringPopulationSize() / (CHR_PER_BLOCK) + 1;
          Blocks.z = 1;


          CalculateKnapsackFintess
                <<<Blocks, Threads>>>
                    (OffspringPopulation->DeviceData, GlobalData.DeviceData);
          CheckAndReportCudaError(__FILE__,__LINE__);



          //----------- Replacement ---------//


          Blocks.x = 1;    
          Blocks.y = (Params->PopulationSize() % (CHR_PER_BLOCK)  == 0) ?
                            Params->PopulationSize() / (CHR_PER_BLOCK)  :
                            Params->PopulationSize() / (CHR_PER_BLOCK) + 1;
          Blocks.z = 1;



          ReplacementKernel
                  <<<Blocks, Threads>>>
                  (MasterPopulation->DeviceData, OffspringPopulation->DeviceData, GetSeed());
         CheckAndReportCudaError(__FILE__,__LINE__);

         
          if (performedEvaluations % Params->StatisticsInterval() == 0){
              GPUStatistics->Calculate(MasterPopulation, Params->GetPrintBest());
             
            /*   printf("Evaluations %6d, MaxFitness %6f, MinFitness %6f, AvgFitness %6f, Diver %6f \n", 
              performedEvaluations, GPUStatistics->HostData->MaxFitness, GPUStatistics->HostData->MinFitness,
                                      GPUStatistics->HostData->AvgFitness, GPUStatistics->HostData->Divergence);
               */
              //if (Params->GetPrintBest())  printf("%s\n", GPUStatistics->GetBestIndividualStr(GlobalData.HostData).c_str());
              outputFile << performedEvaluations << " " << GPUStatistics->HostData->MaxFitness << " ";
              outputFile << GPUStatistics->HostData->AvgFitness << " " << GPUStatistics->HostData->MinFitness << endl;
          }                 
    }
                  
        GPUStatistics->Calculate(MasterPopulation, true);
        outputFile << performedEvaluations << " " << GPUStatistics->HostData->MaxFitness << " ";
        outputFile << GPUStatistics->HostData->AvgFitness << " " << GPUStatistics->HostData->MinFitness << endl;
        outputFile.close();
    
}// end of RunEvolutionCycle
//------------------------------------------------------------------------------