/* 
 * File:        GPU_Evolution.cu
 * Author:      Jiri Jaros
 * Affiliation: Brno University of Technology
 *              Faculty of Information Technology
 *              
 *              and
 * 
 *              The Australian National University
 *              ANU College of Engineering & Computer Science
 *
 * Email:       jarosjir@fit.vutbr.cz
 * Web:         www.fit.vutbr.cz/~jarosjir
 * 
 * Comments:    Implementation file of the GA evolution
 *              This class controls the evolution process on multicore CPU
 *
 * 
 * License:     This source code is distribute under OpenSouce GNU GPL license
 *                
 *              If using this code, please consider citation of related papers
 *              at http://www.fit.vutbr.cz/~jarosjir/pubs.php        
 *      
 *
 * 
 * Created on 30 March 2012, 00:00 PM
 */

#include <iostream>
#include <stdio.h>
#include <sys/time.h>

#include "GPU_Evolution.h"
#include "GPU_Statistics.h"
#include "CUDA_Kernels.h"
#include "Parameters.h"

using namespace std;


//----------------------------------------------------------------------------//
//                              Definitions                                   //
//----------------------------------------------------------------------------//


//----------------------------------------------------------------------------//
//                              Implementation                                //
//                              public methods                                //
//----------------------------------------------------------------------------//

/*
 * Constructor of the class
 */
TGPU_Evolution::TGPU_Evolution(){
    
    // Create parammeter class
    Params    = TParameters::GetInstance();
    pDeviceIdx = 0;
    
    // Select device 
    hipSetDevice(pDeviceIdx);
    
    // Get parameters of the device
    hipDeviceProp_t 	prop;	
    hipGetDeviceProperties (&prop, pDeviceIdx);
   
    pMultiprocessorCount = prop.multiProcessorCount;
    Params->SetGPU_SM_Count(prop.multiProcessorCount);
    
    
    // Load data from file
    GlobalData.LoadFromFile();
    
    // Create populations on GPU
    MasterPopulation    = new TGPU_Population(Params->PopulationSize(), Params->ChromosomeSize());    
    OffspringPopulation = new TGPU_Population(Params->OffspringPopulationSize(), Params->ChromosomeSize());
    
    MigrationPopulation_In  = new TGPU_Population(Params->EmigrantCount(), Params->ChromosomeSize());
    MigrationPopulation_Out = new TGPU_Population(Params->EmigrantCount(), Params->ChromosomeSize());
        
    // Create statistics
    GPUStatistics = new TGPU_Statistics();

    
    pActGeneration = 0;    
    
    // Initialize Random seed
    InitSeed();
    
}// end of TGPU_Evolution
//------------------------------------------------------------------------------
  

/*
 * Destructor of the class
 */
TGPU_Evolution::~TGPU_Evolution(){
    
    delete MasterPopulation;    
    delete OffspringPopulation;
    
    delete MigrationPopulation_In;
    delete MigrationPopulation_Out;
        
    delete GPUStatistics;

    
}// end of Destructor
//------------------------------------------------------------------------------

/*
 * Run Evolution
 */
void TGPU_Evolution::Run(){
    
    Initialize();
            
    RunEvolutionCycle();        
    
}// end of Run
//------------------------------------------------------------------------------


//----------------------------------------------------------------------------//
//                              Implementation                                //
//                              protected methods                             //
//----------------------------------------------------------------------------//


/*
 * Initialize seed
 */
void TGPU_Evolution::InitSeed() {

  struct timeval tp1;  
  
  gettimeofday(&tp1, NULL);
  
  pSeed = (tp1.tv_sec / (pDeviceIdx+1)) * tp1.tv_usec;
  
};// end of InitSeed
//------------------------------------------------------------------------------
    



/*
 * Initialization of the GA
 */
void TGPU_Evolution::Initialize(){
    
        
    pActGeneration = 0;
        
    // Store parameters on GPU and print them out
    Params->StoreParamsOnGPU();
    Params->PrintAllParameters();
    
    // Set elements count
       
    int Elements = Params->ChromosomeSize() * Params->PopulationSize();
                            
    //-- Initialize population --//
    FirstPopulationGenerationKernel
            <<<pMultiprocessorCount * 2, BLOCK_SIZE>>>
            (MasterPopulation->DeviceData, GetSeed());

    dim3 Blocks; 
    dim3 Threads;
            
    Blocks.x = 1;    
    Blocks.y = (Params->PopulationSize() / (CHR_PER_BLOCK) +1);
    Blocks.z = 1;
    
    
    Threads.x = WARP_SIZE;
    Threads.y = CHR_PER_BLOCK;
    Threads.z = 1;
    
        
    
    // Calculate Knapsack fintess
    CalculateKnapsackFintess
            <<<Blocks, Threads>>>
                (MasterPopulation->DeviceData, GlobalData.DeviceData);
    
     
}// end of TGPU_Evolution
//------------------------------------------------------------------------------




/*
 * Run evolutionary cycle for defined number of generations
 * 
 */
void TGPU_Evolution::RunEvolutionCycle(){
    
    
    dim3 Blocks;
    dim3 Threads;
    
    Threads.x = WARP_SIZE;
    Threads.y = CHR_PER_BLOCK;
    Threads.z = 1;
    
    
    
    // Evaluate generations
    for (pActGeneration = 1; pActGeneration < Params->NumOfGenerations(); pActGeneration++) {
      
      
          //-------------Selection -----------//
          Blocks.x = 1;    
          Blocks.y = (Params->OffspringPopulationSize() % (CHR_PER_BLOCK << 1)  == 0) ?
                            Params->OffspringPopulationSize() / (CHR_PER_BLOCK << 1)  :
                            Params->OffspringPopulationSize() / (CHR_PER_BLOCK << 1) + 1;

          Blocks.z = 1;

          GeneticManipulationKernel
                  <<<Blocks, Threads>>>
                  (MasterPopulation->DeviceData, OffspringPopulation->DeviceData, GetSeed());



          //----------- Evaluation ---------//

          Blocks.x = 1;    
          Blocks.y = (Params->OffspringPopulationSize() % (CHR_PER_BLOCK)  == 0) ?
                            Params->OffspringPopulationSize() / (CHR_PER_BLOCK)  :
                            Params->OffspringPopulationSize() / (CHR_PER_BLOCK) + 1;
          Blocks.z = 1;


          CalculateKnapsackFintess
                <<<Blocks, Threads>>>
                    (OffspringPopulation->DeviceData, GlobalData.DeviceData);




          //----------- Replacement ---------//


          Blocks.x = 1;    
          Blocks.y = (Params->PopulationSize() % (CHR_PER_BLOCK)  == 0) ?
                            Params->PopulationSize() / (CHR_PER_BLOCK)  :
                            Params->PopulationSize() / (CHR_PER_BLOCK) + 1;
          Blocks.z = 1;



          ReplacementKernel
                  <<<Blocks, Threads>>>
                  (MasterPopulation->DeviceData, OffspringPopulation->DeviceData, GetSeed());

         
          if (pActGeneration % Params->StatisticsInterval() == 0){
              GPUStatistics->Calculate(MasterPopulation, Params->GetPrintBest());
             
              printf("Generation %6d, MaxFitness %6f, MinFitness %6f, AvgFitness %6f, Diver %6f \n", 
                      pActGeneration, GPUStatistics->HostData->MaxFitness, GPUStatistics->HostData->MinFitness,
                                      GPUStatistics->HostData->AvgFitness, GPUStatistics->HostData->Divergence);
              
              if (Params->GetPrintBest())  printf("%s\n", GPUStatistics->GetBestIndividualStr(GlobalData.HostData).c_str());
          }
                  
              
                  
    }
        
    
    
                  
        GPUStatistics->Calculate(MasterPopulation, true);
        printf("------------------------------------------------------------------------------\n");
        printf("FinalMaxFitness %6f, FinalMinFitness %6f, FinalAvgFitness %6f, FinalDiver %6f \n", 
                GPUStatistics->HostData->MaxFitness, GPUStatistics->HostData->MinFitness,
                GPUStatistics->HostData->AvgFitness, GPUStatistics->HostData->Divergence);
        printf("%s\n", GPUStatistics->GetBestIndividualStr(GlobalData.HostData).c_str());
          
    
}// end of RunEvolutionCycle
//------------------------------------------------------------------------------